#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <string.h>
#include <time.h>

using std::mt19937 ;
using std::random_device;
using std::uniform_real_distribution;

using namespace std;

int Lqs;

hipError_t MuvTheVect(int *h_vect, int *h_aux, int N, int c);

void write_solution(int n, double t, int car, int muvs)
{
    FILE *fp = fopen("Datos.txt", "a+");
    fprintf(fp, "%d, %f, %d, %d\n", n, t,car,muvs);
    fclose(fp);
}

__host__
void prVect(int *vect, int N)
{
    for (int i = 0; i != N; i++)
        printf("%d ", vect[i]);
}

__host__
void FillVect(int *d_vect, hiprandState *globalState, int n)
{
    int ret;
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_int_distribution<int> dist(1.0, 9.0);
    
    for(int i = 0; i != n; i++)
    {
        if (dist(mt) > 5.0)
            ret = 1;
        else
            ret = 0;
        d_vect[i] = ret;
    }
}

__global__ void muvKernel(int *d_vect, int *aux, int N)
{
    int tid = blockIdx.x;
    if (tid < N-1)
    {
        d_vect[tid] = aux[tid] + aux[tid + 1]* -2;
    }
    else
    {
        d_vect[N-1] = aux[N-1] + aux[0]* -2;
    }
}

int main(int argc, char* argv[])
{
    
    int n = stoi(argv[1]);
    int size = n * sizeof(int);
    
    // Hosts
    int *h_vect, *h_aux;
    
    h_vect = (int*)malloc(size);
    h_aux = (int*)malloc(size);
     
    // Llenar
    hiprandState *devState;
    hipMalloc((void **) &devState, n * sizeof(devState));
    
    FillVect(h_vect, devState, n);
    
    int cont = 0;
    for (int i = 0; i != n; i++)
    {
        h_aux[i] = h_vect[i];
        if (h_vect[i] == 1)
            cont++;
    }
     
    MuvTheVect(h_vect, h_aux, n, cont);
        
    free(h_vect); 
    free(h_aux);

    return 0;
}

hipError_t MuvTheVect(int *h_vect, int *h_aux, int N, int c)
{
    
    hipError_t error = hipSuccess;

    int *d_vect;
    int *d_aux;
    
    error = hipMalloc((void**)&d_vect, N * sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error solicitando memoria en la GPU para d_vect\n");
        exit(-1);
    }

    error = hipMalloc((void**)&d_aux, N* sizeof(int));
    if (error != hipSuccess)
    {
        printf("Error solicitando memoria en la GPU para d_aux\n");
        exit(-1);
    }
    
    error = hipMemcpy(d_vect, h_vect, N * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
    }
    
    error = hipMemcpy(d_aux, h_aux, N * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
    }
    
    double start_time = clock();
    muvKernel<<<N, 1>>>(d_vect, d_aux, N);
    
    hipMemcpy(h_vect, d_vect, N*sizeof(int), hipMemcpyDeviceToHost);

    int cont = 0;
    for (int i = 0; i != N; i++)
    {
        h_aux[i] = h_vect[i];
        if (h_vect[i] == 1)
            cont++;
    }
        
    for (int i = 0; i != N; i++)
    {
        if(i != 0)
        {
            if( (h_aux[i-1] == 0) && ((h_aux[i] == 0) || (h_aux[i] == -2)) )
            {
                h_vect[i] = 0;
            }
            else
            {
                if( (h_aux[i-1] == 1) && ((h_aux[i] == 0) || (h_aux[i] == -2)) )
                {
                    h_vect[i] = 1;
                }
                else
                {
                    if (h_aux[i] == 1)
                    {
                        h_vect[i] = 0;
                    }
                    else
                    {
                        h_vect[i] = 1;
                    }
                }
                
            }
            
        }
        else
        {
            if( (h_aux[N-1] == 0) && ((h_aux[i] == 0) || (h_aux[i] == -2)) )
            {
                h_vect[i] = 0;
            }
            else
            {
                if( (h_aux[N-1] == 1) && ((h_aux[i] == 0) || (h_aux[i] == -2)) )
                {
                    h_vect[i] = 1;
                }
                else
                {
                    if (h_aux[i] == 1)
                    {
                        h_vect[i] = 0;
                    }
                    else
                    {
                        h_vect[i] = 1;
                    }
                }
            }
        }
    }
    double time = (clock() - start_time)/CLOCKS_PER_SEC ;
    write_solution(N, time, c, cont);
    
    hipFree(d_vect);
    hipFree(d_aux);
    
    return error;
}